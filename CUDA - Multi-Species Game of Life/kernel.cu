#include "hip/hip_runtime.h"
#define GLEW_STATIC
#include <GL\glew.h>
#include <GLFW\glfw3.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include "cuda_gl_interop.h"
#include <time.h>

#include <stdio.h>
#include <iostream>


#define WIDTH 1024
#define HEIGHT 768
#define LIFE_RATE 70
#define MAX_COLORS 10
#define MAX_FRAMES 30

struct Cell {
	bool alive[10] = { false };
};

GLubyte colorList[11*3] = {
	 255, 0, 0 ,
	 0, 255, 0 ,
	 0, 0, 255 ,
	 1, 255, 254 ,
	 255, 166, 254 ,
	 255, 219, 102 ,
	 0, 100, 1 ,
	 149, 0, 58 ,
	 255, 0, 246 ,
	 255, 147, 126 ,
	 254, 137, 0 
};

int numSpecies = 0;

using namespace std;

hipError_t gameOfLife();
void tickGame(GLubyte *rgb, Cell *device_cell, Cell *device_result_cell, GLubyte *device_colorList);
bool initializeOpenGL();
bool cleanUp();
void render(GLubyte *rgb);
void initializeGame(Cell *device_cell, Cell *device_result_cell);

GLFWwindow* window = 0x00;
GLfloat point_size = 3.0f;

GLubyte *rgb;
GLuint TEX, PBO;
hipGraphicsResource *CUDA_PBO;

int main()
{
	hipError_t cudaStatus = gameOfLife();
	cleanUp();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

hipError_t gameOfLife() {
	hipError_t cudaStatus;
	GLubyte *device_colorList;
	

	Cell *device_cell, *device_result_cell;
	initializeOpenGL();
	// Choose which GPU to run on, change this on a multi-GPU system.
	try {
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			throw - 1;
		}

		// Allocate GPU buffers for Cell Grids  .
		cudaStatus = hipMalloc((void**)&device_cell, WIDTH*HEIGHT * sizeof(Cell));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw - 1;
		}

		cudaStatus = hipMalloc((void**)&device_result_cell, WIDTH*HEIGHT * sizeof(Cell));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw -1;
		}

		initializeGame(device_cell, device_result_cell);

		hipMalloc((void**)&device_colorList, 11 * 3 * sizeof(GLubyte));

		hipMemcpy(device_colorList, colorList, 11 * 3 * sizeof(GLubyte), hipMemcpyHostToDevice);

		glfwSetTime(0);
		int nFrames = 0;
		while (!glfwWindowShouldClose(window)) {
			nFrames++;
			tickGame(rgb, device_cell, device_result_cell, device_colorList);
			render(rgb);
			Cell *tmp = device_cell;
			device_cell = device_result_cell;
			device_result_cell = tmp;
			if (1.0 <= glfwGetTime()) {
				cout << nFrames << " fps" << endl;
				nFrames = 0;
				glfwSetTime(0);
			}
		}

		hipFree(device_colorList);
		hipFree(device_cell);
		hipFree(device_result_cell);
		delete[] rgb;
	}
	catch (int e) {
		hipFree(device_cell);
		hipFree(device_result_cell);
	}
	return cudaStatus;
}

__global__ void computeInteration(int numSpecies, GLubyte *rgb, Cell *cellGrid, Cell *copyCell, GLubyte *species) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	rgb[i*(WIDTH * 3) + (j * 3) + 0] = 0;
	rgb[i*(WIDTH * 3) + (j * 3) + 1] = 0;
	rgb[i*(WIDTH * 3) + (j * 3) + 2] = 0;
	for (int currentSpecies = 0; currentSpecies < numSpecies; currentSpecies++) {
		if (cellGrid[i*WIDTH + j].alive != NULL && cellGrid[i*WIDTH + j].alive[currentSpecies]) {
			rgb[i*(WIDTH * 3) + (j * 3) + 0] += species[(currentSpecies * 3) + 0];
			rgb[i*(WIDTH * 3) + (j * 3) + 1] += species[(currentSpecies * 3) + 1];
			rgb[i*(WIDTH * 3) + (j * 3) + 2] += species[(currentSpecies * 3) + 2];
		}
		int numberAliveNeighbors = 0;
		for (int iN = -1; iN <= 1; iN++)
			for (int jN = -1; jN <= 1; jN++)
				if (i != 0 && i != HEIGHT - 1 && j != 0 && j != WIDTH - 1 && !(iN == 0 && jN == 0) && cellGrid[((i + iN)*WIDTH) + (j + jN)].alive[currentSpecies])
					numberAliveNeighbors++;
		if (numberAliveNeighbors < 2 || numberAliveNeighbors > 3)
			copyCell[i*WIDTH + j].alive[currentSpecies] = false;
		else if (cellGrid[i*WIDTH + j].alive[currentSpecies] || (!cellGrid[i*WIDTH + j].alive[currentSpecies] && numberAliveNeighbors == 3))
			copyCell[i*WIDTH + j].alive[currentSpecies] = true;
		else {
			copyCell[i*WIDTH + j].alive[currentSpecies] = false;
		}
	}
}

void tickGame(GLubyte *rgb, Cell *device_cell, Cell *device_result_cell, GLubyte *device_colorList) {
	hipGraphicsMapResources(1, &CUDA_PBO, 0);
	GLubyte *device_rgb;
	size_t num_bytes = WIDTH*HEIGHT*3;
	hipGraphicsResourceGetMappedPointer((void**)&device_rgb,
		&num_bytes, CUDA_PBO);

	
	dim3  grid(128, 64);
	dim3 block(WIDTH / grid.x, HEIGHT / grid.y);
	computeInteration << <grid, block >> > (numSpecies, device_rgb, device_cell, device_result_cell, device_colorList);
	hipDeviceSynchronize();
	hipError_t cudaStatus;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		throw - 1;
	}
	hipGraphicsUnmapResources(1, &CUDA_PBO, 0);
}

__global__ void initialState(int numSpecies, Cell *cellGrid, Cell *copyCell)
{

	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(clock(), 0, 0, &state);

	int rand1 = (hiprand(&state) % numSpecies);
	int rand2 = hiprand(&state) % 100;

	int speciesID = rand1;
	if (rand2 <= LIFE_RATE) {
		cellGrid[j*WIDTH + i].alive[speciesID] = true;
	}else
		cellGrid[j*WIDTH + i].alive[speciesID] = false;
}

void initializeGame(Cell *device_cell, Cell *device_result_cell) {
	dim3  grid(512, 256);
	dim3 block(WIDTH/ grid.x, HEIGHT/ grid.y);
	hipError_t cudaStatus;
	std::cout << "Indicate the number of species (Recommended 5 - 10): ";
	std::cin >> numSpecies;
	initialState <<<grid, block >>>(numSpecies, device_cell, device_result_cell);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		throw - 1;
	}
}

bool initializeOpenGL() {
	if (!glfwInit()) {
		fprintf(stderr, "ERROR: could not start GLFW3\n");
		return false;
	}
	glfwWindowHint(GLFW_DOUBLEBUFFER, GL_TRUE);
	window = glfwCreateWindow(WIDTH, HEIGHT, "Multi-Species Game", NULL, NULL);
	int w, h;
	glfwGetWindowSize(window, &w, &h);
	glViewport(0, 0, w, h);
	glfwMakeContextCurrent(window);
	glewExperimental = GL_TRUE;
	glewInit();
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LESS);


	rgb = new GLubyte[WIDTH * HEIGHT*3];

	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &TEX);
	glBindTexture(GL_TEXTURE_2D, TEX);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, WIDTH, HEIGHT, 0, GL_RGB,
		GL_UNSIGNED_BYTE, rgb);

	glGenBuffers(1, &PBO);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, PBO);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, WIDTH * HEIGHT * 3 * sizeof(GLubyte),
		rgb, GL_STREAM_COPY);

	hipError_t result = hipGraphicsGLRegisterBuffer(&CUDA_PBO, PBO,
		cudaGraphicsMapFlagsWriteDiscard);
	return result == hipSuccess;
}

bool cleanUp() {
	glfwTerminate();
	return true;
}

void draw() {
	glBindTexture(GL_TEXTURE_2D, TEX);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, PBO);

	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, WIDTH, HEIGHT,
		GL_RGB, GL_UNSIGNED_BYTE, 0);

	glBegin(GL_QUADS);
	glTexCoord2f(0, 0);
	glVertex2f(-1, -1);
	glTexCoord2f(1, 0);
	glVertex2f(1, -1);
	glTexCoord2f(1, 1);
	glVertex2f(1, 1);
	glTexCoord2f(0, 1);
	glVertex2f(-1, 1);
	glEnd();

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glBindTexture(GL_TEXTURE_2D, 0);
}

void render(GLubyte *rgb) {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glClearColor(0.1f, 0.2f, 0.2f, 1.0f);
	glPointSize(point_size);
	draw();
	glfwPollEvents();
	glfwSwapBuffers(window);
}
